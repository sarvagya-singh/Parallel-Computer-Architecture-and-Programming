#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 6 
#define BLOCK_SIZE 4 

__global__ void matrixMulKernel(int *A, int *B, int *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

void printMatrix(int *matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", matrix[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main() {
    int size = N * N * sizeof(int);
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() % 10;
        h_B[i] = rand() % 10;
    }

    printf("Matrix A:\n");
    printMatrix(h_A, N);
    printf("Matrix B:\n");
    printMatrix(h_B, N);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrixMulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix C:\n");
    printMatrix(h_C, N);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
