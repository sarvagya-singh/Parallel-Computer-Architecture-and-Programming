#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

__global__ void calculateSine(float *input, float *output, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        output[idx] = sinf(input[idx]);
    }
}

int main() {
    int size = 5;
    float h_input[] = {0.0f, 0.5f, 1.0f, 1.5708f, 3.1416f};
    float h_output[size];

    float *d_input, *d_output;
    
    hipMalloc((void**)&d_input, size * sizeof(float));
    hipMalloc((void**)&d_output, size * sizeof(float));
    
    hipMemcpy(d_input, h_input, size * sizeof(float), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    calculateSine<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size);

    hipMemcpy(h_output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);
 
    printf("Input angles (in radians):\n");
    for (int i = 0; i < size; i++) {
        printf("%f ", h_input[i]);
    }
    printf("\nSine values:\n");
    for (int i = 0; i < size; i++) {
        printf("%f ", h_output[i]);
    }
    printf("\n");

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
