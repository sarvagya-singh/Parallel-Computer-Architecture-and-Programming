#include <hip/hip_runtime.h>
#include <stdio.h>

#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void cudaCount(char* A, unsigned int *d_count) {
    int i = threadIdx.x;
    if (A[i] == 'a')
        atomicAdd(d_count, 1);
}

int main() {
    char A[N];
    char* d_A;
    unsigned int count = 0, result = 0;  
    unsigned int *d_count;

    printf("Enter a String : \n");
    fgets(A, N, stdin);
    A[strcspn(A, "\n")] = '\0';  
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc((void**)&d_A, strlen(A) * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(unsigned int));

    hipMemcpy(d_A, A, strlen(A) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error1: %s\n", hipGetErrorString(error));
    }

    cudaCount<<<1, strlen(A)>>>(d_A, d_count);

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error2: %s\n", hipGetErrorString(error));
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);

    hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("Total occurrences of 'a': %u\n", result);
    printf("Time Taken: %f ms\n", elapsed_time);

    hipFree(d_A);
    hipFree(d_count);

    return 0;
}
