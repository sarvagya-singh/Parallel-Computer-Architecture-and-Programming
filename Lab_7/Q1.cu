#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MAX_SENTENCE 1024
#define MAX_WORD 100

__global__ void countWord(const char* sentence, const char* word, int sentenceLength, int wordLength, unsigned int* d_count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i > sentenceLength - wordLength)
        return;
    
    bool match = true;
    for (int j = 0; j < wordLength; j++) {
        if (sentence[i + j] != word[j]) {
            match = false;
            break;
        }
    }
    
    if (match) {
        atomicAdd(d_count, 1);
    }
}

int main() {
    char sentence[MAX_SENTENCE];
    char word[MAX_WORD];
    
    printf("Enter a sentence:\n");
    fgets(sentence, MAX_SENTENCE, stdin);

    printf("Enter the word to search for:\n");
    fgets(word, MAX_WORD, stdin);

    int sentenceLength = strlen(sentence);
    int wordLength = strlen(word);

    if (sentenceLength < wordLength) {
        printf("The word is longer than the sentence. Occurrences: 0\n");
        return 0;
    }
    
    char *d_sentence, *d_word;
    unsigned int *d_count;
    unsigned int count = 0;
    
    hipMalloc((void**)&d_sentence, sentenceLength * sizeof(char));
    hipMalloc((void**)&d_word, wordLength * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(unsigned int));
    
    hipMemcpy(d_sentence, sentence, sentenceLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, wordLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocks = (sentenceLength - wordLength + 1 + threadsPerBlock - 1) / threadsPerBlock;
    
    countWord<<<blocks, threadsPerBlock>>>(d_sentence, d_word, sentenceLength, wordLength, d_count);
    
    hipDeviceSynchronize();
    
    hipMemcpy(&count, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    printf("The word \"%s\" appears %u times in the sentence.\n", word, count);
    
    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_count);
    
    return 0;
}
